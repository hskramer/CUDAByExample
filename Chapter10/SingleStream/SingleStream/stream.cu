#include "hip/hip_runtime.h"

#include <book.h>

#define N   (1024 * 1024)
#define FULL_DATA_SIZE   (N * 20)


__global__ void kernel(int *a, int *b, int *c)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < N)
	{
		int idx1 = (idx + 1) % 256;
		int idx2 = (idx + 2) % 256;
		float   as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
		float   bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
		c[idx] = (as + bs) / 2;
	}
}


int main(void)
{
	hipEvent_t     start, stop;
	float           elapsedTime;

	hipStream_t    stream;
	int *h_a, *h_b, *h_c;
	int *d_a, *d_b, *d_c;

	// start the timers
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));

	// initialize the stream
	HANDLE_ERROR(hipStreamCreate(&stream));

	// allocate the memory on the GPU
	HANDLE_ERROR(hipMalloc((void**)&d_a, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&d_b, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&d_c, N * sizeof(int)));

	// allocate host locked memory, used to stream
	HANDLE_ERROR(hipHostAlloc((void**)&h_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**)&h_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**)&h_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault));

	for (int i = 0; i<FULL_DATA_SIZE; i++)
	{
		h_a[i] = rand();
		h_b[i] = rand();
	}

	HANDLE_ERROR(hipEventRecord(start, 0));
	// now loop over full data, in bite-sized chunks
	for (int i = 0; i<FULL_DATA_SIZE; i += N)
	{
		// copy the locked memory to the device, async
		HANDLE_ERROR(hipMemcpyAsync(d_a, h_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream));
		HANDLE_ERROR(hipMemcpyAsync(d_b, h_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream));

		kernel << <N / 256, 256, 0, stream >> >(d_a, d_b, d_c);

		// copy the data from device to locked memory
		HANDLE_ERROR(hipMemcpyAsync(h_c + i, d_c, N * sizeof(int), hipMemcpyDeviceToHost, stream));

	}
	// copy result chunk from locked to full buffer
	HANDLE_ERROR(hipStreamSynchronize(stream));

	HANDLE_ERROR(hipEventRecord(stop, 0));

	HANDLE_ERROR(hipEventSynchronize(stop));
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));

	printf("Time taken:  %3.2f ms\n", elapsedTime);

	// cleanup the streams and memory
	HANDLE_ERROR(hipHostFree(h_a));
	HANDLE_ERROR(hipHostFree(h_b));
	HANDLE_ERROR(hipHostFree(h_c));
	HANDLE_ERROR(hipFree(d_a));
	HANDLE_ERROR(hipFree(d_b));
	HANDLE_ERROR(hipFree(d_c));
	HANDLE_ERROR(hipStreamDestroy(stream));

	return 0;
}

