#include "hip/hip_runtime.h"
#include <stdio.h>
#include <book.h>

#define N	50000

__global__
void add(int *a, int *b, int *c)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < N)
	{
		c[i] = a[i] + b[i];
		i += blockDim.x*gridDim.x;
	}
}
int main(void)
{
	int a[N], b[N], c[N];
	int *d_a, *d_b, *d_c;

	HANDLE_ERROR(hipMalloc((void**)&d_a, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&d_b, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&d_c, N * sizeof(int)));

	for (int i = 0; i < N; i++)
	{
		a[i] = i;
		b[i] = i * i;
	}

	HANDLE_ERROR(hipMemcpy(d_a, a, N * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_b, b, N * sizeof(int), hipMemcpyHostToDevice));

	add <<<256, 256>> > (d_a, d_b, d_c);

	HANDLE_ERROR(hipMemcpy(c, d_c, N * sizeof(int), hipMemcpyDeviceToHost));

	bool success = true;
	for (int i = 0; i < N; i++)
	{
		if ((a[i] + b[i]) != c[i])
		{
			printf_s("Error: %d + %d != %d", a[i], b[i], c[i]);
			success = false;
		}
	}

	if (success)
	{
		for (int i = 0; i < 1000; i++)
		{
			printf_s("%d + %d = %d\n", b[i], a[i], c[i]);
		}
	}

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;

}