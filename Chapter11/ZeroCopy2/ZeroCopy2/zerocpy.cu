#include <book.h>



int main()
{

	float           *a, *b, c, *partial_c;
	float           *d_a, *d_b, *d_partial_c;
	float           elapsedTime;

	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));

	// allocate the memory on the CPU
	HANDLE_ERROR(hipHostAlloc((void**)&a, size * sizeof(float), hipHostMallocWriteCombined | hipHostMallocMapped));
	HANDLE_ERROR(hipHostAlloc((void**)&b, size * sizeof(float), hipHostMallocWriteCombined | hipHostMallocMapped));
	HANDLE_ERROR(hipHostAlloc((void**)&partial_c, blocksPerGrid * sizeof(float), hipHostMallocMapped));

	// find out the GPU pointers
	HANDLE_ERROR(hipHostGetDevicePointer(&d_a, a, 0));
	HANDLE_ERROR(hipHostGetDevicePointer(&d_b, b, 0));
	HANDLE_ERROR(hipHostGetDevicePointer(&d_partial_c, partial_c, 0));

	// fill in the host memory with data
	for (int i = 0; i < size; i++)
	{
		a[i] = i;
		b[i] = i * 2;
	}

	HANDLE_ERROR(hipEventRecord(start, 0));

	dot << <blocksPerGrid, threadsPerBlock >> >(size, d_a, d_b, d_partial_c);

	HANDLE_ERROR(hipDeviceSynchronize());
	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));

	// finish up on the CPU side
	c = 0;
	for (int i = 0; i < blocksPerGrid; i++)
	{
		c += partial_c[i];
	}

	HANDLE_ERROR(hipHostFree(a));
	HANDLE_ERROR(hipHostFree(b));
	HANDLE_ERROR(hipHostFree(partial_c));

	const int num_streams = 8;

	hipStream_t streams[num_streams];
	float *data[num_streams];

	for (int i = 0; i < num_streams; i++)
	{
		hipStreamCreate(&streams[i]);

		hipMalloc(&data[i], N * sizeof(float));

		// launch one worker kernel per stream
		kernel << <1, 64, 0, streams[i] >> >(data[i], N);

		// launch a dummy kernel on the default stream
		kernel << <1, 1 >> >(0, 0);
	}

	hipDeviceReset();

	return 0;

}