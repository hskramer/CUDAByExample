#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <book.h>

#define SIZE    (100*1024*1024)

__global__ void histo_kernel(unsigned char *buffer,	long size, unsigned int *histo) 
{
	// calculate the starting index and the offset to the next
	// block that each thread will be processing
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	while (i < size) 
	{
		atomicAdd(&histo[buffer[i]], 1);
		i += stride;
	}
}

int main(void) 
{
	unsigned char *buffer =	(unsigned char*)big_random_block(SIZE);

	// capture the start time
	// starting the timer here so that we include the cost of
	// all of the operations on the GPU.
	hipEvent_t     start, stop;
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipEventRecord(start, 0));

	// allocate memory on the GPU for the file's data
	unsigned char *d_buffer;
	unsigned int *d_histo;
	HANDLE_ERROR(hipMalloc((void**)&d_buffer, SIZE));
	HANDLE_ERROR(hipMemcpy(d_buffer, buffer, SIZE, hipMemcpyHostToDevice));

	HANDLE_ERROR(hipMalloc((void**)&d_histo, 256 * sizeof(int)));
	HANDLE_ERROR(hipMemset(d_histo, 0, 256 * sizeof(int)));

	// kernel launch - 2x the number of mps gave best timing
	hipDeviceProp_t  prop;
	HANDLE_ERROR(hipGetDeviceProperties(&prop, 0));
	int blocks = prop.multiProcessorCount;
	histo_kernel <<<blocks * 16, 256>>>(d_buffer, SIZE, d_histo);

	unsigned int    histo[256];
	HANDLE_ERROR(hipMemcpy(histo, d_histo, 256 * sizeof(int), hipMemcpyDeviceToHost));

	// get stop time, and display the timing results
	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));

	float   elapsedTime;
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));

	printf("Time to generate:  %3.1f ms\n", elapsedTime);

	long histoCount = 0;
	for (int i = 0; i<256; i++)
	{
		histoCount += histo[i];
	}
	printf("Histogram Sum:  %ld\n", histoCount);

	// verify that we have the same counts via CPU
	for (int i = 0; i<SIZE; i++)
		histo[buffer[i]]--;

	for (int i = 0; i<256; i++) 
	{
		if (histo[i] != 0)
			printf("Failure at %d!  Off by %d\n", i, histo[i]);
	}

	HANDLE_ERROR(hipEventDestroy(start));
	HANDLE_ERROR(hipEventDestroy(stop));
	hipFree(d_histo);
	hipFree(d_buffer);
	free(buffer);
	return 0;
}
