#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <book.h>
#include <cpu_anim.h>

#define DIM	1024
#define PI 3.1415926535897932f
#define MAX_TEMP 1.0f
#define MIN_TEMP 0.0001f
#define SPEED	0.25f

__global__ void copy_const_kernel(float *iptr, const float *cptr)
{
	// map from threadIdx/blockIdx to pixel position
	int	x = blockIdx.x * blockDim.x + threadIdx.x;
	int	y = blockIdx.y * blockDim.y + threadIdx.y;
	int offset = x + y *blockDim.x*gridDim.x;

	if (cptr[offset] != 0)	iptr[offset] = cptr[offset];
}

__global__ void blend_kernel(float *outSrc,  float *inSrc)
{
	// map from threadIdx/blockIdx to pixel position
	int	x = blockIdx.x * blockDim.x + threadIdx.x;
	int	y = blockIdx.y * blockDim.y + threadIdx.y;
	int offset = x + y * blockDim.x * gridDim.x;

	int	left = offset - 1;
	int right = offset + 1;
	if (x == 0)		left++;
	if (x == DIM - 1)	  right--;

	int top = offset - DIM;
	int bottom = offset + DIM;
	if (y == 0)		top += DIM;
	if (y == DIM - 1) bottom -= DIM;

	outSrc[offset] = inSrc[offset] + SPEED * (inSrc[top] + inSrc[bottom] + inSrc[left] + inSrc[right] - inSrc[offset] * 4);
}

struct DataBlock {
	unsigned char	*output_bitmap;
	float			*d_inSrc;
	float			*d_outSrc;
	float			*d_constSrc;
	CPUAnimBitmap	*bitmap;
	hipEvent_t		start, stop;
	float			totalTime;
	float			frames;
};

void anim_gpu(DataBlock *d, int ticks)
{
	HANDLE_ERROR(hipEventRecord(d->start, 0));
	dim3	blocks(DIM / 32, DIM / 32);
	dim3	threads(32, 32);
	CPUAnimBitmap	*bitmap = d->bitmap;

	for (int i = 0; i < 90; i++)
	{
		copy_const_kernel <<<blocks, threads >>> (d->d_inSrc, d->d_constSrc);

		blend_kernel << <blocks, threads >> > (d->d_outSrc, d->d_inSrc);

		swap(d->d_inSrc, d->d_outSrc);
	}

	float_to_color <<<blocks, threads >>> (d->output_bitmap, d->d_inSrc);

	HANDLE_ERROR(hipMemcpy(bitmap->get_ptr(), d->output_bitmap, bitmap->image_size(), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipEventRecord(d->stop, 0));
	HANDLE_ERROR(hipEventSynchronize(d->stop));
	
	float	elapsedTime;

	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, d->start, d->stop));

	d->totalTime += elapsedTime;
	++d->frames;

	printf("Average time per frame:   %3.2f ms\n", d->totalTime / d->frames);



}

void anim_exit(DataBlock *d)
{
	hipFree(d->d_inSrc);
	hipFree(d->d_outSrc);
	hipFree(d->d_constSrc);
	
	HANDLE_ERROR(hipEventDestroy(d->start));
	HANDLE_ERROR(hipEventDestroy(d->stop));
}

int main(void)
{
	DataBlock		data;
	CPUAnimBitmap	bitmap(DIM, DIM, &data);
	data.bitmap = &bitmap;
	data.totalTime = 0;
	data.frames = 0;

	HANDLE_ERROR(hipEventCreate(&data.start));
	HANDLE_ERROR(hipEventCreate(&data.stop));

	HANDLE_ERROR(hipMalloc((void**)&data.output_bitmap, bitmap.image_size()));

	// assume float == 4 chars in size (ie rgba)
	HANDLE_ERROR(hipMalloc((void**)&data.d_inSrc, bitmap.image_size()));
	HANDLE_ERROR(hipMalloc((void**)&data.d_outSrc, bitmap.image_size()));
	HANDLE_ERROR(hipMalloc((void**)&data.d_constSrc, bitmap.image_size()));

	// intialize the constant data
	float *temp = (float*)malloc(bitmap.image_size());

	for (int i = 0; i < DIM*DIM; i++)
	{
		temp[i] = 0;
		int x = i % DIM;
		int y = i / DIM;
		if ((x > 300) && (x < 600) && (y > 310) && (y < 601))
			temp[i] = MAX_TEMP;
	}
	temp[DIM * 100 + 100] = (MAX_TEMP + MIN_TEMP) / 2;
	temp[DIM * 700 + 100] = MIN_TEMP;
	temp[DIM * 300 + 300] = MIN_TEMP;
	temp[DIM * 200 + 700] = MIN_TEMP;
	for (int y = 800; y < 900; y++)
	{
		for (int x = 400; x < 500; x++)
		{
			temp[x + y*DIM] = MIN_TEMP;
		}
	}

	HANDLE_ERROR(hipMemcpy(data.d_constSrc, temp,bitmap.image_size(),hipMemcpyHostToDevice));

	// initialize the input data
	for (int y = 800; y<DIM; y++)
	{
		for (int x = 0; x<200; x++)
		{
			temp[x + y*DIM] = MAX_TEMP;
		}
	}

	HANDLE_ERROR(hipMemcpy(data.d_inSrc, temp,bitmap.image_size(),hipMemcpyHostToDevice));
	free(temp);

	bitmap.anim_and_exit((void(*)(void*, int))anim_gpu,	(void(*)(void*))anim_exit);

	return 0;
}